#include "hip/hip_runtime.h"
#include "viewer.cuh"
#include <thrust/device_vector.h>
#include "hip/hip_vector_types.h"
#include <hip/hip_runtime.h>
#include <vector_functions.h>

template <typename T>
__device__ T mix(T a, T b, float interp) {
	return (1 - interp) * a + interp * b;
}

__device__ float interp_tsdf_diff(const float3& pos, const float3& vol_start, const float3& voxel, const int3& vol_dim, float *tsdf_diff) {
	float3 idx = (pos - vol_start) / voxel;
	int3 floored_idx = make_int3(floorf(idx.x), floorf(idx.y), floorf(idx.z));
	float3 frac_idx = idx - make_float3(floored_idx.x, floored_idx.y, floored_idx.z);
	int base_idx = vol_dim.y * vol_dim.z * floored_idx.x + vol_dim.z * floored_idx.y + floored_idx.z;
	float diffs[8];
	for (uint8_t i = 0; i < 2; ++i)
	{
		for (uint8_t j = 0; j < 2; ++j) 
		{
			for (uint8_t k = 0; k < 2; ++k)
			{
				int vol_idx = base_idx + vol_dim.y * vol_dim.z * i + vol_dim.z * j + k;
				diffs[i * 4 + j * 2 + k] = tsdf_diff[vol_idx];
			}
		}
	}
	float low = mix(mix(diffs[0], diffs[4], frac_idx.x), mix(diffs[2], diffs[6], frac_idx.x), frac_idx.y);
	float high = mix(mix(diffs[1], diffs[5], frac_idx.x), mix(diffs[3], diffs[7], frac_idx.x), frac_idx.y);
	return mix(low, high, frac_idx.z);
}

__device__ uchar3 interp_tsdf_color(const float3& pos, const float3& vol_start, const float3& voxel, const int3& vol_dim, uchar3 *tsdf_color) {
	float3 idx = (pos - vol_start) / voxel;
	int3 floored_idx = make_int3(floorf(idx.x), floorf(idx.y), floorf(idx.z));
	float3 frac_idx = idx - make_float3(floored_idx.x, floored_idx.y, floored_idx.z);
	int base_idx = vol_dim.y * vol_dim.z * floored_idx.x + vol_dim.z * floored_idx.y + floored_idx.z;
	float3 colors[8];
	for (uint8_t i = 0; i < 2; ++i)
	{
		for (uint8_t j = 0; j < 2; ++j)
		{
			for (uint8_t k = 0; k < 2; ++k)
			{
				int vol_idx = base_idx + vol_dim.y * vol_dim.z * i + vol_dim.z * j + k;
				colors[i * 4 + j * 2 + k] = make_float3(tsdf_color[vol_idx].x, tsdf_color[vol_idx].y, tsdf_color[vol_idx].z);
			}
		}
	}
	float3 low = mix(mix(colors[0], colors[4], frac_idx.x), mix(colors[2], colors[6], frac_idx.x), frac_idx.y);
	float3 high = mix(mix(colors[1], colors[5], frac_idx.x), mix(colors[3], colors[7], frac_idx.x), frac_idx.y);
	float3 res = mix(low, high, frac_idx.z);
	return make_uchar3(res.x, res.y, res.z);
}

__global__ void show_tsdf_kernel(float *s2w, float3 *c, float3 *vol_start, float3 *vol_end, float3 *voxel,
	int3 *vol_dim, float *tsdf_diff, uchar3 *tsdf_color,
	int width, int height, uchar3 *output)
{
	uint16_t idx_x = blockIdx.x * blockDim.x + threadIdx.x;
	uint16_t idx_y = blockIdx.y * blockDim.y + threadIdx.y;

	float4 screen_pos = make_float4(idx_x, idx_y, 1.f, 1.f);
	float3 target = make_float3(dot(make_float4(s2w[0], s2w[1], s2w[2], s2w[3]), screen_pos),
		dot(make_float4(s2w[4], s2w[5], s2w[6], s2w[7]), screen_pos),
		dot(make_float4(s2w[8], s2w[9], s2w[10], s2w[11]), screen_pos)
	);

	float3 d = normalize(target - c[0]);
	float3 inv_d = 1.f / d;
	float3 tbot = inv_d * (vol_start[0] - c[0]);
	float3 ttop = inv_d * (vol_end[0] - c[0]);

	float3 tmin = make_float3(min(ttop.x, tbot.x), min(ttop.y, tbot.y), min(ttop.z, tbot.z));
	float tnear = max(max(tmin.x, tmin.y), tmin.z);
	tnear = max(tnear, 0.01f);

	float3 tmax = make_float3(max(ttop.x, tbot.x), max(ttop.y, tbot.y), max(ttop.z, tbot.z));
	float tfar = min(min(tmax.x, tmax.y), tmax.z);
	tfar = min(tfar, 100.f);
	if (tnear > tfar) return;

	float t = tnear;
	float f_tt = 0;
	float stepsize = voxel[0].x;
	float f_t = interp_tsdf_diff(c[0] + t * d, vol_start[0], voxel[0], vol_dim[0], tsdf_diff);
	if (f_t > 0) {
		for (; t < tfar; t += stepsize)
		{
			f_tt = interp_tsdf_diff(c[0] + t * d, vol_start[0], voxel[0], vol_dim[0], tsdf_diff);
			if (f_tt < 0.f)
			{
				break;
			}
			if (f_tt < voxel[0].x / 2.f)
			{
				stepsize = voxel[0].x / 4.f;
			}
			f_t = f_tt;
		}
		if (f_tt < 0.f)
		{
			t += stepsize * f_tt / (f_t - f_tt);
			output[(idx_y * width + idx_x)] = interp_tsdf_color(c[0] + t * d, vol_start[0], voxel[0], vol_dim[0], tsdf_color);
		}
	}
}


template<typename T>
T* malloc_and_cpy(T *host_ptr, size_t cnt) {
	T *device_ptr;
	hipMalloc(&device_ptr, cnt * sizeof(T));
	hipMemcpy(device_ptr, host_ptr, cnt * sizeof(T), hipMemcpyHostToDevice);
	return device_ptr;
}

cv::Mat show_tsdf(const TSDF& tsdf, int width, int height, float angle, float dist) {
	cv::Mat img(height, width, CV_8UC3);

	float rot[16] = { std::cosf(angle), 0, -std::sinf(angle), dist * std::sinf(angle), 0, 1, 0, 0, std::sinf(angle), 0, std::cosf(angle), dist - dist * std::cosf(angle), 0, 0, 0, 1 };
	cv::Mat extrinsic(4, 4, CV_32F, rot);
	cv::Mat s2w = extrinsic * tsdf.get_intrinsic_inv();

	float center[3] = { 0 };
	center[0] = (dist + 0.5f) * std::sinf(angle);
	center[2] = (dist + 0.5f) - (dist + 0.5f) * std::cosf(angle);

	auto vol_dim = tsdf.get_dim();
	int size = vol_dim[0] * vol_dim[1] * vol_dim[2];

	float *s2w_d = malloc_and_cpy((float*)s2w.data, 16);
	float *c_d = malloc_and_cpy((float*)center, 3);
	float *tsdf_diff_d = malloc_and_cpy((float*)tsdf.get_tsdf_diff(), size);
	uchar3 *tsdf_color_d = malloc_and_cpy((uchar3*)tsdf.get_tsdf_color(), size);
	float *vol_start_d = malloc_and_cpy((float*)tsdf.get_vol_start().val, 3);
	float *vol_end_d = malloc_and_cpy((float*)tsdf.get_vol_end().val, 3);
	int *vol_dim_d = malloc_and_cpy((int*)vol_dim.val, 3);
	float *voxel_d = malloc_and_cpy((float*)tsdf.get_voxel().val, 3);
	uchar3 *output_d = malloc_and_cpy((uchar3*)img.data, width * height);

	show_tsdf_kernel << <dim3((width - 1) / 32 + 1, (height - 1) / 32 + 1, 1), dim3(32, 32, 1) >> > (
		s2w_d,
		(float3*)c_d,
		(float3*)vol_start_d,
		(float3*)vol_end_d,
		(float3*)voxel_d,
		(int3*)vol_dim_d,
		tsdf_diff_d,
		tsdf_color_d,
		width,
		height,
		output_d
		);
	hipMemcpy(img.data, output_d, width * height * sizeof(uchar3), hipMemcpyDeviceToHost);
	hipFree(s2w_d);
	hipFree(c_d);
	hipFree(vol_start_d);
	hipFree(vol_end_d);
	hipFree(voxel_d);
	hipFree(vol_dim_d);
	hipFree(tsdf_diff_d);
	hipFree(tsdf_color_d);
	hipFree(output_d);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		std::stringstream strstr;
		strstr << "run_kernel launch failed" << std::endl;
		strstr << hipGetErrorString(error);
		throw strstr.str();
	}
	cv::imshow("img", img);
	cv::waitKey(1);
	return img;
}