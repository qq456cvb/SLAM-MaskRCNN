#include "hip/hip_runtime.h"
#include "viewer.cuh"
#include <thrust/device_vector.h>
#include "hip/hip_vector_types.h"
#include <hip/hip_runtime.h>
#include <vector_functions.h>
#include "utils.cuh"

template<typename T>
T* malloc_and_cpy(T *host_ptr, size_t cnt) {
	T *device_ptr;
	hipMalloc(&device_ptr, cnt * sizeof(T));
	hipMemcpy(device_ptr, host_ptr, cnt * sizeof(T), hipMemcpyHostToDevice);
	return device_ptr;
}

__global__ void show_tsdf_kernel(float *s2w, float3 *c, float3 *vol_start, float3 *vol_end, float3 *voxel,
	int3 *vol_dim, float *tsdf_diff, uchar3 *tsdf_color, uint32_t *tsdf_cnt,
	int width, int height, uchar3 *output)
{
	uint16_t idx_x = blockIdx.x * blockDim.x + threadIdx.x;
	uint16_t idx_y = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx_x >= width) return;
	if (idx_y >= height) return;

	float4 screen_pos = make_float4(idx_x, idx_y, 1.f, 1.f);
	float3 target = make_float3(dot(make_float4(s2w[0], s2w[1], s2w[2], s2w[3]), screen_pos),
		dot(make_float4(s2w[4], s2w[5], s2w[6], s2w[7]), screen_pos),
		dot(make_float4(s2w[8], s2w[9], s2w[10], s2w[11]), screen_pos)
	);

	float3 d = normalize(target - c[0]);
	float3 inv_d = 1.f / d;
	float3 tbot = inv_d * (vol_start[0] - c[0]);
	float3 ttop = inv_d * (vol_end[0] - c[0]);

	float3 tmin = make_float3(min(ttop.x, tbot.x), min(ttop.y, tbot.y), min(ttop.z, tbot.z));
	float tnear = max(max(tmin.x, tmin.y), tmin.z);
	tnear = max(tnear, 0.01f);

	float3 tmax = make_float3(max(ttop.x, tbot.x), max(ttop.y, tbot.y), max(ttop.z, tbot.z));
	float tfar = min(min(tmax.x, tmax.y), tmax.z);
	tfar = min(tfar, 100.f);
	if (tnear > tfar) return;

	float t = tnear;
	float f_tt = 0;
	float stepsize = voxel[0].x;
	float f_t = interp_tsdf_diff(c[0] + t * d, vol_start[0], voxel[0], vol_dim[0], tsdf_diff);
	if (f_t > 0) {
		for (; t < tfar; t += stepsize)
		{
			f_tt = interp_tsdf_diff(c[0] + t * d, vol_start[0], voxel[0], vol_dim[0], tsdf_diff);
			if (f_tt < 0.f)
			{
				break;
			}
			if (f_tt < voxel[0].x / 2.f)
			{
				stepsize = voxel[0].x / 4.f;
			}
			f_t = f_tt;
		}
		if (f_tt < 0.f)
		{
			t += stepsize * f_tt / (f_t - f_tt);
			//output[(idx_y * width + idx_x)] = interp_tsdf_color(c[0] + t * d, vol_start[0], voxel[0], vol_dim[0], tsdf_color);
			float cnts[MAX_OBJECTS];
			interp_tsdf_cnt(c[0] + t * d, vol_start[0], voxel[0], vol_dim[0], tsdf_cnt, cnts);
			float max_cnt = 0;
			uint8_t obj_idx = 0;
			for (uint8_t k = 0; k < MAX_OBJECTS; k++)
			{
				if (cnts[k] > max_cnt) {
					max_cnt = cnts[k];
					obj_idx = k;
				}
			}
			output[(idx_y * width + idx_x)] = make_uchar3(obj_idx * 20, obj_idx * 20, obj_idx * 20);
		}
	}
}


cv::Mat show_tsdf(const TSDF& tsdf, int width, int height, float angle, float dist) {
	cv::Mat img(height, width, CV_8UC3, cv::Scalar(0));

	float rot[16] = { std::cosf(angle), 0, -std::sinf(angle), dist * std::sinf(angle), 0, 1, 0, 0, std::sinf(angle), 0, std::cosf(angle), dist - dist * std::cosf(angle), 0, 0, 0, 1 };
	cv::Mat extrinsic(4, 4, CV_32F, rot);
	cv::Mat s2w = extrinsic * tsdf.get_intrinsic_inv();

	float center[3] = { 0 };
	center[0] = (dist + 0.5f) * std::sinf(angle);
	center[2] = (dist + 0.5f) - (dist + 0.5f) * std::cosf(angle);

	auto vol_dim = tsdf.get_dim();
	int size = vol_dim[0] * vol_dim[1] * vol_dim[2];

	float *s2w_d = malloc_and_cpy((float*)s2w.data, 16);
	float *c_d = malloc_and_cpy((float*)center, 3);
	float *tsdf_diff_d = malloc_and_cpy((float*)tsdf.get_tsdf_diff(), size);
	uchar3 *tsdf_color_d = malloc_and_cpy((uchar3*)tsdf.get_tsdf_color(), size);
	uint32_t *tsdf_cnt_d = malloc_and_cpy((uint32_t*)tsdf.get_tsdf_cnt(), size * MAX_OBJECTS);
	float *vol_start_d = malloc_and_cpy((float*)tsdf.get_vol_start().val, 3);
	float *vol_end_d = malloc_and_cpy((float*)tsdf.get_vol_end().val, 3);
	int *vol_dim_d = malloc_and_cpy((int*)vol_dim.val, 3);
	float *voxel_d = malloc_and_cpy((float*)tsdf.get_voxel().val, 3);
	uchar3 *output_d = malloc_and_cpy((uchar3*)img.data, width * height);

	show_tsdf_kernel << <dim3((width - 1) / 32 + 1, (height - 1) / 32 + 1, 1), dim3(32, 32, 1) >> > (
		s2w_d,
		(float3*)c_d,
		(float3*)vol_start_d,
		(float3*)vol_end_d,
		(float3*)voxel_d,
		(int3*)vol_dim_d,
		tsdf_diff_d,
		tsdf_color_d,
		tsdf_cnt_d,
		width,
		height,
		output_d
		);
	hipMemcpy(img.data, output_d, width * height * sizeof(uchar3), hipMemcpyDeviceToHost);

	hipFree(s2w_d);
	hipFree(c_d);
	hipFree(vol_start_d);
	hipFree(vol_end_d);
	hipFree(voxel_d);
	hipFree(vol_dim_d);
	hipFree(tsdf_diff_d);
	hipFree(tsdf_color_d);
	hipFree(tsdf_cnt_d);
	hipFree(output_d);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		std::stringstream strstr;
		strstr << "run_kernel launch failed" << std::endl;
		strstr << hipGetErrorString(error);
		throw strstr.str();
	}
	cv::imshow("img", img);
	cv::waitKey();
	return img;
}